#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define forward 0
#define backward 1

typedef struct _Node_info{
	u_short parent_index;
	u_int potential_flow;
} Node_info;

__global__ void find_augmenting_path(u_short* residual_capacity, Node_info* node_info, bool* frontier, bool* visited, u_int total_nodes, u_int sink){

	int node_id = blockIdx.x * blockDim.x + threadIdx.x;
	if(!frontier[sink] && node_id < total_nodes && frontier[node_id]){

		frontier[node_id] = false;
		visited[node_id] = true;

		Node_info neighbour;
		u_int capacity;

		for(int i=0; i< total_nodes; i++){

			if(frontier[i] || visited[i] || (capacity = residual_capacity[node_id * total_nodes + i] <= 0)){
				continue;
			}

			frontier[i] = true;

			neighbour = node_info[i];
			neighbour.parent_index = node_id;
			neighbour.potential_flow = min(node_info[node_id].potential_flow, capacity);
		}
	}
}

__global__ void reset(Node_info* node_info, bool* frontier, bool* visited, int source, int total_nodes){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < total_nodes){
		frontier[id] = id == source;
		visited[id] = false;
		Node_info current_node_info = node_info[id];
		current_node_info.potential_flow = UINT_MAX;
	}
}

bool is_frontier_empty_or_sink_found(bool* frontier, int N, int sink_pos){
	for (int i = N-1; i > -1; --i) {
		if(frontier[i]){
			return i == sink_pos;
		}
	}
	return true;
}

int main(){

	unsigned int N = 4;
	unsigned int source=0, sink=N-1;
	unsigned int current_vertex, bottleneck_flow;
	Node_info current_node_info;

	unsigned int max_flow = 0;

	u_short *residual_capacity, *d_residual_capacity;

	char edge_info_matrix[N][N];

	bool* frontier, *visited;
	bool* d_frontier, *d_visited;

	Node_info* node_info;
	Node_info* d_node_info;

	size_t matrix_size = N * N * sizeof(u_short);
	residual_capacity = (u_short *)malloc(matrix_size);

	//size_t edge_info_size = N * N * sizeof(char);
	//edge_info_matrix = (char *)malloc(edge_info_size);

	size_t node_infos_size = N * sizeof(Node_info);
	node_info = (Node_info*)malloc(node_infos_size);

	size_t vertices_size = N * sizeof(bool);
	frontier = (bool *)malloc(vertices_size);
	visited = (bool *)malloc(vertices_size);

	for (int i = 0; i < N; ++i) {
		frontier[i] = false;
		visited[i] = false;

		for (int j = 0; j < N; ++j) {
			edge_info_matrix[i][j] = forward;
			residual_capacity[i * N + j] = 0;
		}
	}

	residual_capacity[0 * N + 1] = 3;
	residual_capacity[1 * N + 3] = 2;
	residual_capacity[1 * N + 2] = 1;
	residual_capacity[0 * N + 2] = 2;
	residual_capacity[2 * N + 3] = 3;

	edge_info_matrix[1][0] = backward;
	edge_info_matrix[3][1] = backward;
	edge_info_matrix[2][1] = backward;
	edge_info_matrix[2][0] = backward;
	edge_info_matrix[3][2] = backward;

	frontier[0] = true;

	hipMalloc((u_short **)&d_residual_capacity, matrix_size);
	hipMalloc((Node_info**)&d_node_info,node_infos_size);
	hipMalloc((bool **)&d_frontier, vertices_size);
	hipMalloc((bool **)&d_visited, vertices_size);

	hipMemcpy(d_residual_capacity, residual_capacity, matrix_size, hipMemcpyHostToDevice);
	hipMemcpy(d_node_info, node_info, node_infos_size, hipMemcpyHostToDevice);
	hipMemcpy(d_frontier, frontier, vertices_size, hipMemcpyHostToDevice);
	hipMemcpy(d_visited, visited, matrix_size, hipMemcpyHostToDevice);

	bool found_augmenting_path;

	int threads = 256;
	int blocks = ceil(N * 1.0 /threads);

	do{

		while(!is_frontier_empty_or_sink_found(frontier, N,sink)){
				printf("is_frontier_empty_or_sink_found");
				// Invoke kernel
				find_augmenting_path<<< blocks, threads >>>(d_residual_capacity, d_node_info, d_frontier, d_visited, N, sink);

				// Copy back frontier from device
				hipMemcpy(frontier, d_frontier, vertices_size, hipMemcpyDeviceToHost);
		}

		found_augmenting_path = frontier[sink];

		if(!found_augmenting_path){
			break;
		}

		// copy node_info from device to host
		hipMemcpy(node_info, d_node_info, node_infos_size, hipMemcpyDeviceToHost);

		bottleneck_flow = node_info[sink].potential_flow;
		max_flow += bottleneck_flow;
		printf("maxflow -%d\n", max_flow);

		//bool edge_info_changed = false;

		for(current_vertex = sink; current_vertex != source; current_vertex = current_node_info.parent_index){
			current_node_info = node_info[sink];

			if(edge_info_matrix[current_node_info.parent_index][current_vertex] == forward){
				residual_capacity[current_node_info.parent_index * N + current_vertex] -= bottleneck_flow;

//				if(edge_info_matrix[current_node_info.parent_index][current_vertex] != backward){
//					edge_info_matrix[current_node_info.parent_index][current_vertex] = backward;
//					edge_info_changed = true;
//				}
				residual_capacity[current_vertex * N + current_node_info.parent_index] += bottleneck_flow;
			}else{
				residual_capacity[current_vertex * N + current_node_info.parent_index] -= bottleneck_flow;
				residual_capacity[current_node_info.parent_index * N + current_vertex] += bottleneck_flow;
			}
		}

		// copy residual_capacity, edge_info to device
		hipMemcpy(d_residual_capacity, residual_capacity, matrix_size, hipMemcpyHostToDevice);

//		if(edge_info_changed){
//			cudaMemcpy(d_edge_info_matrix, edge_info_matrix, edge_info_size, cudaMemcpyHostToDevice);
//		}

		// reset visited, frontier, node_info
		reset<<<blocks, threads >>>(d_node_info,frontier, visited, source, N);

	}while(found_augmenting_path);

	free(edge_info_matrix);
	free(residual_capacity);
	free(frontier);
	free(visited);
	free(node_info);

	hipFree(d_residual_capacity);
	hipFree(d_node_info);
	hipFree(d_frontier);
	hipFree(d_visited);

	return 0;
}
