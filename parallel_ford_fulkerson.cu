
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#define milliseconds 1e3

using namespace std;

typedef struct _Node_info{
	u_short parent_index;
	u_int potential_flow;
} Node_info;

void readInput(const char* filename, u_int total_nodes, u_short* residual_capacity) {

	ifstream file;
	file.open(filename);

	if (!file) {
        cout <<  "Error reading file!";
        exit(1);
    }

    string line;
    u_int source, destination;
    u_short capacity;

    while (file) {

        getline(file, line);

        if (line.empty()) {
            continue;
        }

        std::stringstream linestream(line);
        linestream >> source >> destination >> capacity;
        residual_capacity[source * total_nodes + destination] = capacity;
    }

    file.close();
}

__global__ void find_augmenting_path(u_short* residual_capacity, Node_info* node_info, bool* frontier, bool* visited, 
	u_int total_nodes, u_int sink, u_int* locks){

	int node_id = blockIdx.x * blockDim.x + threadIdx.x;

	if(!frontier[sink] && node_id < total_nodes && frontier[node_id]){

		frontier[node_id] = false;
		visited[node_id] = true;

		Node_info *neighbour;
		Node_info current_node_info = node_info[node_id];
		u_int capacity, i, count = 0;
		
		while(++count < total_nodes){
			i = (node_id+count) % total_nodes;	

			if(frontier[i] || visited[i] || ((capacity = residual_capacity[node_id * total_nodes + i]) <= 0)){
				continue;
			}

			if(atomicCAS(locks+i, 0 , 1) == 1 || frontier[i]){
				continue;
			}

			frontier[i] = true;
			locks[i] = 0;

			neighbour = node_info + i;
			neighbour->parent_index = node_id;
			neighbour->potential_flow =  min(current_node_info.potential_flow, capacity);
		}
	}
}

__global__ void reset(Node_info* node_info, bool* frontier, bool* visited, int source, int total_nodes, u_int* locks){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < total_nodes){
		frontier[id] = id == source;
		visited[id] = false;
		node_info[id].potential_flow = UINT_MAX;
		locks[id] = 0; 
	}
}

void reset_host(bool* frontier, int source, int total_nodes){
	for (int i = 0; i < total_nodes; i++) {
		frontier[i] = i == source;
	}
}

bool is_frontier_empty_or_sink_found(bool* frontier, int N, int sink_pos){
	for (int i = N-1; i > -1; --i) {
		if(frontier[i]){
			return i == sink_pos;
		}
	}
	return true;
}

int main(int argc, char** argv){

	if(argc < 3){
		printf("Specify filename & number of vertices\n");
		return 1;
	}

	u_int N = atoi(argv[2]);
	u_short *residual_capacity;

	size_t matrix_size = N * N * sizeof(u_short);
	residual_capacity = (u_short *)malloc(matrix_size);
	memset(residual_capacity, 0, matrix_size); 

	readInput(argv[1], N, residual_capacity);

	u_int source=0, sink=N-1;
	u_int current_vertex, bottleneck_flow;
	u_int max_flow = 0;

	Node_info* current_node_info;
	u_short* d_residual_capacity;
	u_int* d_locks;
	bool* frontier, *visited;
	bool* d_frontier, *d_visited;

	Node_info* node_info;
	Node_info* d_node_info;

    clock_t start_time = clock(); 

	size_t node_infos_size = N * sizeof(Node_info);
	node_info = (Node_info*)malloc(node_infos_size);

	size_t vertices_size = N * sizeof(bool);
	frontier = (bool *)malloc(vertices_size);
	visited = (bool *)malloc(vertices_size);

	for (int i = 0; i < N; ++i) {
		frontier[i] = false;
		visited[i] = false;

		node_info[i].potential_flow = UINT_MAX;
	}

	frontier[0] = true;

	size_t locks_size = N * sizeof(u_int);

	hipMalloc((void **)&d_residual_capacity, matrix_size);
	hipMalloc((void **)&d_locks, locks_size);
	hipMalloc((void **)&d_node_info,node_infos_size);
	hipMalloc((void **)&d_frontier, vertices_size);
	hipMalloc((void **)&d_visited, vertices_size);

	hipMemcpy(d_residual_capacity, residual_capacity, matrix_size, hipMemcpyHostToDevice);
	hipMemcpy(d_node_info, node_info, node_infos_size, hipMemcpyHostToDevice);
	hipMemcpy(d_frontier, frontier, vertices_size, hipMemcpyHostToDevice);
	hipMemcpy(d_visited, visited, vertices_size, hipMemcpyHostToDevice);

	bool found_augmenting_path;

	int threads = 256;
	int blocks = ceil(N * 1.0 /threads);

	do{

		// reset visited, frontier, node_info, locks
		reset<<<blocks, threads >>>(d_node_info, d_frontier, d_visited, source, N, d_locks);
		reset_host(frontier, source, N);

		while(!is_frontier_empty_or_sink_found(frontier, N, sink)){
				// Invoke kernel
				find_augmenting_path<<< blocks, threads >>>(d_residual_capacity, d_node_info, d_frontier, d_visited, N, sink, d_locks);

				// Copy back frontier from device
				hipMemcpy(frontier, d_frontier, vertices_size, hipMemcpyDeviceToHost);
		}

		found_augmenting_path = frontier[sink];

		if(!found_augmenting_path){
			break;
		}

		// copy node_info from device to host
		hipMemcpy(node_info, d_node_info, node_infos_size, hipMemcpyDeviceToHost);

		bottleneck_flow = node_info[sink].potential_flow;
		max_flow += bottleneck_flow;

		for(current_vertex = sink; current_vertex != source; current_vertex = current_node_info->parent_index){
			current_node_info = node_info + current_vertex;
			residual_capacity[current_node_info->parent_index * N + current_vertex] -= bottleneck_flow;
			residual_capacity[current_vertex * N + current_node_info->parent_index] += bottleneck_flow;
		}

		// copy residual_capacity, edge_info to device
		hipMemcpy(d_residual_capacity, residual_capacity, matrix_size, hipMemcpyHostToDevice);

	}while(found_augmenting_path);

	cout << "\nmaxflow " << max_flow << endl;
    double time_taken = ((double)clock() - start_time)/CLOCKS_PER_SEC * milliseconds; // in milliseconds 
	cout << time_taken << " ms for thread size- " << threads << endl;

	free(residual_capacity);
	free(frontier);
	free(visited);
	free(node_info);

	hipFree(d_residual_capacity);
	hipFree(d_node_info);
	hipFree(d_frontier);
	hipFree(d_visited);

	return 0;
}
